#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;

__global__ void BinaryCUDA(unsigned char* Image, int Channels, int umbral) {
	int x = blockIdx.x;
    int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;
	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
		if (Image[idx+i] > umbral) {
			Image[idx + i] = 255;
		}
		else {
			Image[idx + i] = 0;
		}
	}
}

void ImageBinary(unsigned char* Input_Image, int Height, int Width, int Channels, int umbral) {
	unsigned char* Dev_Input_Image = NULL;

	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	BinaryCUDA<<<Grid_Image, 8>>>(Dev_Input_Image, Channels, umbral);

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}

__global__ void BinaryInvCUDA(unsigned char* Image, int Channels, int umbral) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
		if (Image[idx + i] > umbral) {
			Image[idx + i] = 0;
		}
		else {
			Image[idx + i] = 255;
		}
	}
}

void ImageBinaryInv(unsigned char* Input_Image, int Height, int Width, int Channels, int umbral) {
	unsigned char* Dev_Input_Image = NULL;

	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	BinaryInvCUDA <<<Grid_Image, 8>>>(Dev_Input_Image, Channels, umbral);

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}

__global__ void TruncCUDA(unsigned char* Image, int Channels, int umbral) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
		if (Image[idx + i] > umbral) {
			Image[idx + i] = Image[idx + i];
		}
		else {
			Image[idx + i] = 255;
		}
	}
}

void ImageTrunc(unsigned char* Input_Image, int Height, int Width, int Channels, int umbral) {
	unsigned char* Dev_Input_Image = NULL;

	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	TruncCUDA<<<Grid_Image, 8>>>(Dev_Input_Image, Channels, umbral);

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}

__global__ void TozeroCUDA(unsigned char* Image, int Channels, int umbral) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		if (Image[idx + i] > umbral) {
			Image[idx + i] = 0;
		}
		else {
			Image[idx + i] = Image[idx + i];
		}
	}
}

void ImageTozero(unsigned char* Input_Image, int Height, int Width, int Channels, int umbral) {
	unsigned char* Dev_Input_Image = NULL;

	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	TruncCUDA<<<Grid_Image, 8>>>(Dev_Input_Image, Channels, umbral);

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}


__global__ void TozeroInvCUDA(unsigned char* Image, int Channels, int umbral) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x+y*gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		if (Image[idx + i] > umbral) {
			Image[idx + i] = Image[idx + i];
		}
		else {
			Image[idx + i] = 0;
		}
	}
}

void ImageTozeroInv(unsigned char* Input_Image, int Height, int Width, int Channels, int umbral) {
	unsigned char* Dev_Input_Image = NULL;


	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	TozeroInvCUDA<<<Grid_Image, 8>>>(Dev_Input_Image, Channels, umbral);

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}

int main(int argc, char* argv[])
{
	Mat image = imread(argv[1], CV_LOAD_IMAGE_COLOR);
	int umbral = 112;

	if (!image.data)
	{
		printf("Could not open or find the image\n");
		return -1;
    }
    Mat output_image = image.clone();

    string option = argv[2];

	int o = 0;

	if (option == "binary") {
		o = 1;
	}
	else if (option == "binaryInv") {
		o = 2;
	}
	else if (option == "trunc") {
		o = 3;
	}
	else if (option == "tozero") {
		o = 4;
	}
	else if (option == "tozeroInv") {
		o = 5;
	}
	printf("\nImage processing starting\n");
	switch (o) {
		case 1:
			ImageBinary(output_image.data, output_image.cols, output_image.rows, output_image.channels(), umbral);
			break;
		case 2:
			ImageBinaryInv(output_image.data, output_image.cols, output_image.rows, output_image.channels(), umbral);
			break;
		case 3:
			ImageTrunc(output_image.data, output_image.cols, output_image.rows, output_image.channels(), umbral);
			break;
		case 4:
			ImageTozero(output_image.data, output_image.cols, output_image.rows, output_image.channels(), umbral);
			break;
		case 5:
			ImageTozeroInv(output_image.data, output_image.cols, output_image.rows, output_image.channels(), umbral);
			break;
	}

    imwrite(argv[1], output_image);
	printf("\nImage processing done\n");
}